#include "hip/hip_runtime.h"
#include <iostream>
#include "ee155_utils.hxx"
#include "matrix.hxx"
using namespace std;
const int BS = 32;	// The blocks are BS x BS.


///////////////////////////////
// This is the CUDA kernel function for you to write.
//
__global__ void mat_mult (float *d_A, float *d_B, float *d_C, int N) {
    int rb = blockIdx.x;
    int cb = blockIdx.y;
    int ri = threadIdx.x;
    int ci = threadIdx.y;
    
    __shared__ float SA[BS][BS], SB[BS][BS];
    //printf("In thread with r=(%d,%d) c=(%d,%d)\n", rB,rI,cB,cI);
    // Copy the data to shared memory
    for (int kb = 0; kb < gridDim.x; kb++) {
        SA[ri][ci] = d_A[N*(rb*BS+ri)+kb*BS+ci];
        SB[ri][ci] = d_B[N*(kb*BS+ri)+cb*BS+ci];
        __syncthreads();

        // Do actual computations
        for (int ki = 0; ki < BS; ki++) {
            d_C[N*(rb*BS+ri)+cb*BS+ci] += SA[ri][ki] * SB[ki][ci];
        }
        __syncthreads();
    }
    

}



///////////////////////////////
// This is the host function for you to write.
// It allocates memory and moves data between CPU<->GPU
void Matrix::mpy1 (const Matrix &A, const Matrix &B, int BS) {

    // Copy A from host memory to device memory.
    int numElem=N()*N(), sizeBytes = numElem*4;
    float *d_A = NULL;
    hipError_t err = hipMalloc((void **)&d_A, sizeBytes);
    ERR_CHK (err, "Failed to allocate device matrix A");

    err = hipMemcpy (d_A, A.data.data(), sizeBytes, hipMemcpyHostToDevice);
    ERR_CHK (err, "Failed to copy matrix A from host to device");

    // Allocate device memory for B.
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, sizeBytes);
    ERR_CHK (err, "Failed to allocate device matrix B");

    err = hipMemcpy (d_B, B.data.data(), sizeBytes, hipMemcpyHostToDevice);
    ERR_CHK (err, "Failed to copy matrix B from host to device");

    // Allocate device memory for C.
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, sizeBytes);
    ERR_CHK (err, "Failed to allocate device matrix C");

    // Set C to all zeroes
    err = hipMemset(d_C, 0, sizeBytes);
    ERR_CHK (err, "Failed to set matrix C to zero");
    int N=A.N();
    int Nb=A.N()/BS;

    dim3 gridSize(Nb, Nb);
    dim3 blockSize(BS, BS);
    // Launch the CUDA Kernel
    mat_mult <<< gridSize, blockSize >>> (d_A, d_B, d_C, N);
    // err = hipGetLastError();
    // ERR_CHK (err, "Failed to launch matrix multiplication kernel");

    // Copy the result from device memory to host memory.
    err = hipMemcpy (data.data(), d_C, sizeBytes, hipMemcpyDeviceToHost);
    ERR_CHK (err, "Failed to copy matrix C from device to host");

    // Free device memory.
    err = hipFree(d_A);
    ERR_CHK (err, "Failed to free CUDA matrix A");
}